#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../common/book.h"

__global__ void vectorAdd(const float *A, const float *B, float *C,
    int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char **argv){

     // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);
    float *h_A, *h_B, *h_C;
    float *dev_a, *dev_b, *dev_c;
    
    // allocate the memory on the CPU
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, size ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, size ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, size ) );

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / static_cast<float>(RAND_MAX);
        h_B[i] = rand() / static_cast<float>(RAND_MAX);
    }

    HANDLE_ERROR(hipMemcpy(dev_a, h_A, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, h_B, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_c, h_C, size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
            threadsPerBlock);
            
    dim3 cudaBlockSize(threadsPerBlock, 1, 1);
    dim3 cudaGridSize(blocksPerGrid, 1, 1);

    vectorAdd<<<cudaGridSize, cudaBlockSize>>>(dev_a, dev_b, dev_c, numElements);

    HANDLE_ERROR(hipMemcpy(h_C, dev_c, size, hipMemcpyDeviceToHost));

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
        fprintf(stderr, "Result verification failed at element %d!\n", i);
        exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    free(h_A);
    free(h_B);
    free(h_C);
    return 0;   
}